#include "hip/hip_runtime.h"
#include "klustakwik.h"
#include "params.h"

#include "hip/hip_runtime.h"  
#include ""

#include <hipblas.h>
#include <hipsolver.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>

#include <stdio.h>
#include <iostream>

#define _USE_MATH_DEFINES
#include <math.h>

#define BLOCKDIM 128


//global kernel
//===========================================compute weight=======================================================//

__global__ void c_Weight(int nClustersAlive, int priorPoint, int nPoints, int NoisePoint, int *d_AliveIndex, int *d_nClassMembers, float *d_Weight) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < nClustersAlive)
	{
		int c = d_AliveIndex[tid];
		if (c>0) d_Weight[c] = ((float)d_nClassMembers[c] + priorPoint) / (nPoints + NoisePoint + priorPoint*(nClustersAlive - 1));
		else d_Weight[c] = ((float)d_nClassMembers[c] + NoisePoint) / (nPoints + NoisePoint + priorPoint*(nClustersAlive - 1));
	}
}
//===========================================compute mean=======================================================//

__global__ void c_nClassMembers(int nPoints, int *d_Class, int *d_nClassMembers) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	if (tidx < nPoints) {
		atomicAdd(&d_nClassMembers[d_Class[tidx]], 1);
	}
}
__global__ void matCopy(int npoints, int ndims, int nDims, int *d_rowId, int *d_colId, float *d_sourceMat, float *d_copyMat) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < ndims && tidy < npoints) {
		int p = d_rowId[tidy];
		int nd = d_colId[tidx];
		//printf("in the kernel,this is the x: %d,y: %d,xx: %d, yy:%d\n", tidx, tidy, p, nd);
		d_copyMat[tidy * ndims + tidx] = d_sourceMat[p * nDims + nd];
	}

}

__global__ void c_CheckDead(int nClustersAlive, int *d_AliveIndex, int *d_nClassMembers, int *d_ClassAlive) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid <= nClustersAlive)
	{
		int c = d_AliveIndex[tid];
		if (c>0 && d_nClassMembers[c]<1)
			d_ClassAlive[c] = 0;
	}
}

__global__ void c_FeatureSum(int nPoints, int nDims, int *d_Class, float *d_Mean, float *d_Data) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < nPoints && tidy < nDims) {
		int c = d_Class[tidx];
		atomicAdd(&d_Mean[c*nDims + tidy], d_Data[tidx*nDims + tidy]);
	}
}

__global__ void c_FeatureMean(int nClustersAlive, int nDims, int *d_AliveIndex, float *d_Mean, int *d_nClassMembers) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < nClustersAlive && tidy < nDims) {
		int c = d_AliveIndex[tidx];
		d_Mean[c*nDims + tidy] /= d_nClassMembers[c];
	}
}

__global__ void c_AllVector2Mean(int nPoints, int nDims, float *d_AllVector2Mean, float* d_Mean, float *d_Data, int *d_Class) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < nPoints && tidy < nDims) {
		int c = d_Class[tidx];
		d_AllVector2Mean[tidx *nDims + tidy] = d_Data[tidx*nDims + tidy] - d_Mean[c*nDims + tidy];
	}
}

__global__ void c_CorrectionTerm(int nDims, int NumUnmasked, int NumMasked, int NumPointsInThisClass,
	int *d_CurrentUnmasked, int *d_CurrentMasked, float *d_cov, float *d_dig,
	int *d_PointsInThisClass, float *d_CorrectionTerm) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < NumUnmasked)
	{
		float ccf = 0.0;
		int i = d_CurrentUnmasked[tid];
		for (int q = 0; q<NumPointsInThisClass; q++)
		{
			const int p = d_PointsInThisClass[q];
			ccf += d_CorrectionTerm[p*nDims + i];
		}
		d_cov[tid*NumUnmasked + tid] += ccf;
	}
	__syncthreads();
	if (tid < NumMasked)
	{
		float ccf = 0.0;
		int i = d_CurrentMasked[tid];
		for (int q = 0; q<NumPointsInThisClass; q++)
		{
			const int p = d_PointsInThisClass[q];
			ccf += d_CorrectionTerm[p*nDims + i];
		}
		d_dig[tid] += ccf;
	}
}

__global__ void addNoiseVar(int NumUnmasked, int NumMasked, int priorPoint, float *d_NoiseVariance,
	int *d_CurrentUnmasked, int *d_CurrentMasked, float *d_cov, float *d_dig) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < NumUnmasked) {
		d_cov[tid * NumUnmasked + tid] += priorPoint * d_NoiseVariance[d_CurrentUnmasked[tid]];
	}
	__syncthreads();
	if (tid < NumMasked) {
		d_dig[tid] += priorPoint * d_NoiseVariance[d_CurrentMasked[tid]];
	}
}
//=============================================================================//
__global__ void checkDigSingular(int nunmasked, int mark, float *d_dig, float *d_invdig) {
	int tid = blockDim.x * blockIdx.x + threadsIdx.x;
	if (tid < nmasked) {
		if (d_dig[tid] <= 0)
		{
			atomicAdd(&mark, 1);
			break;
		}
		else d_invdig[tid] = (float)sqrt(d_dig[tid]);
	}
}

__global__ void makeInvCov(int nunmasked, float *d_cov, float *d_invcov) {
	int tid = blockDim.x * blockIdx.x + threadsIdx.x;
	if (tid < nunmasked)
		d_invcov[tid] = d_cov[tid * nunmasked + tid];
}
__global__ void calSubMahal(int nDims, int pId, float *d_invDig, float *d_CorrectionTerm,
	float *d_subMah) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	if (tidx < nDims)
		d_subMah[tidx] = d_invDig[tidx] * d_CorrectionTerm[pId*nDims + tidx];
}

__global__ void extractPoints2Mean(int nunmasked, int nPoints, int nDims, int clusterId,
	int *d_CurrentUnmasked,
	float *d_Data, float *d_Mean, float *d_points2mean) {
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;
	if (tidx < nunmasked && tidy < nPoints) {
		int nd = d_CurrentUnmasked[tidx];
		d_points2mean[tidy * nunmasked + tidx] =
			d_Data[tidy * nDims + nd] - d_Mean[clusterId * nDims + nd];
	}
}


void KK::MEstep()
{
	clock_t clock1 = clock();

	// clear arrays
	//memset((void*)&nClassMembers.front(), 0, MaxPossibleClusters * sizeof(int));
	//memset((void*)&Mean.front(), 0, MaxPossibleClusters*nDims * sizeof(float));
	if (Debug) { Output("Entering Unmasked Mstep \n"); }

	thrust::device_vector<int> d_nClassMembers(MaxPossibleClusters, 0);
	thrust::device_vector<float>  d_Mean(MaxPossibleClusters*nDims, 0.0);

	thrust::device_vector<int> d_Class = Class;

	// Accumulate total number of points in each class
	int gridd = (nPoints / BLOCKDIM) + 1;
	c_nClassMembers << <gridd, BLOCKDIM >> > (nPoints,
		thrust::raw_pointer_cast(&d_Class[0]),
		thrust::raw_pointer_cast(&d_nClassMembers[0])
		);

	// check for any dead classes
	thrust::device_vector<int> d_AliveIndex = AliveIndex;
	thrust::device_vector<int>  d_ClassAlive = ClassAlive;

	c_CheckDead << <BLOCKDIM, BLOCKDIM >> > (nClustersAlive,
		thrust::raw_pointer_cast(&d_AliveIndex[0]),
		thrust::raw_pointer_cast(&d_nClassMembers[0]),
		thrust::raw_pointer_cast(&d_ClassAlive[0])
		);
	thrust::copy(d_ClassAlive.begin(), d_ClassAlive.end(), ClassAlive.begin());

	Reindex();
	thrust::copy(d_AliveIndex.begin(), d_AliveIndex.end(), AliveIndex.begin());

	// Normalize by total number of points to give class weight
	// Also check for dead classes

	thrust::device_vector<float> d_Weight(MaxPossibleClusters);
	c_Weight << <BLOCKDIM, BLOCKDIM >> > (nClustersAlive, priorPoint, nPoints, NoisePoint,
		thrust::raw_pointer_cast(&d_AliveIndex[0]),
		thrust::raw_pointer_cast(&d_nClassMembers[0]),
		thrust::raw_pointer_cast(&d_Weight[0]));
	thrust::copy(d_Weight.begin(), d_Weight.end(), Weight.begin());


	//================================================compute Cov=======================================//  

	thrust::device_vector<float> d_Data = Data;

	int dimx = 32;
	int dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid(512, 512);

	c_FeatureSum << <grid, block >> > (nPoints, nDims,
		thrust::raw_pointer_cast(&d_Class[0]),
		thrust::raw_pointer_cast(&d_Mean[0]),
		thrust::raw_pointer_cast(&d_Data[0])
		);

	//printf("d_Mean.size: %d\n", d_Mean.size());
	//for (int i = 0; i < 1000; i++) std::cout << d_Mean[i] << "  ";
	//printf("\n\n\n");

	c_FeatureMean << <grid, block >> >(nClustersAlive, nDims,
		thrust::raw_pointer_cast(&d_AliveIndex[0]),
		thrust::raw_pointer_cast(&d_Mean[0]),
		thrust::raw_pointer_cast(&d_nClassMembers[0])
		);

	//printf("d_Mean.size: %d\n", d_Mean.size());
	//for (int i = 0; i < 1000; i++) std::cout << d_Mean[i] << "  ";
	//printf("\n\n\n");

	thrust::device_vector<float> d_AllVector2Mean(nPoints*nDims);
	c_AllVector2Mean << <grid, block >> >(nPoints, nDims,
		thrust::raw_pointer_cast(&d_AllVector2Mean[0]),
		thrust::raw_pointer_cast(&d_Mean[0]),
		thrust::raw_pointer_cast(&d_Data[0]),
		thrust::raw_pointer_cast(&d_Class[0])
		);
	//printf("d_AllVector2Mean.size: %d\n", d_AllVector2Mean.size());
	//for (int i = 0; i <1000; i++) std::cout << d_AllVector2Mean[i] << "  ";
	//printf("\n\n\n");

	vector< vector<int> > PointsInClass(MaxPossibleClusters);
	for (int p = 0; p<nPoints; p++)
	{
		int c = Class[p];
		PointsInClass[c].push_back(p);
	}

	//printf("d_AllVector2Mean.size: %d\n", d_AllVector2Mean.size());
	//for (int i = 0; i < d_AllVector2Mean.size(); i++) std::cout << d_AllVector2Mean[i] << "  ";
	//printf("\n\n\n");

	//E step used
	thrust::copy(d_Mean.begin(), d_Mean.end(), Mean.begin());
	thrust::copy(d_ClassAlive.begin(), d_ClassAlive.end(), ClassAlive.begin());


	thrust::copy(d_AliveIndex.begin(), d_AliveIndex.end(), AliveIndex.begin());
	thrust::copy(d_nClassMembers.begin(), d_nClassMembers.end(), nClassMembers.begin());

	//=======================================================================================
	// Compute the cluster masks, used below to optimise the computation
	ComputeClusterMasks();
	// Empty the dynamic covariance matrices (we will fill it up as we go)
	DynamicCov.clear();

	for (int cc = 0; cc < nClustersAlive; cc++)
	{
		int c = AliveIndex[cc];
		vector<int> &CurrentUnmasked = ClusterUnmaskedFeatures[c];
		vector<int> &CurrentMasked = ClusterMaskedFeatures[c];
		DynamicCov.push_back(BlockPlusDiagonalMatrix(CurrentMasked, CurrentUnmasked));
	}

	thrust::device_vector<float> d_CorrectionTerm = CorrectionTerm;
	thrust::device_vector<float> d_NoiseVariance = NoiseVariance;

	//this is  a  big  kernel!!

	clock_t clock2 = clock();
	//printf("%d    %d\n", nClustersAlive,DynamicCov.size());
	//printf("\n");
	for (int cc = 0; cc < nClustersAlive; cc++)
	{
		const int c = AliveIndex[cc];
		const vector<int> &PointsInThisClass = PointsInClass[c];
		const vector<int> &CurrentUnmasked = ClusterUnmaskedFeatures[c];
		const vector<int> &CurrentMasked = ClusterMaskedFeatures[c];
		BlockPlusDiagonalMatrix &CurrentCov = DynamicCov[cc];

		const int npoints = (int)PointsInThisClass.size();
		const int nunmasked = (int)CurrentUnmasked.size();
		const int nmasked = (int)CurrentMasked.size();

		thrust::device_vector<int> d_PointsInThisClass = PointsInClass[c];
		thrust::device_vector<int> d_CurrentUnmasked = ClusterUnmaskedFeatures[c];
		thrust::device_vector<int> d_CurrentMasked = ClusterMaskedFeatures[c];
		thrust::device_vector<float> d_cov(nunmasked * nunmasked, 0.0);
		thrust::device_vector<float> d_dig(nmasked, 0.0);
		/*
		printf("%d    %d  \n", npoints)
		for (int q = 0; q < npoints; q++)
		{
		const int p = PointsInThisClass[q];
		const float * __restrict av2mp = &(AllVector2Mean[p*nDims]);
		for (int ii = 0; ii < nDims; ii++)
		{
		printf("%f  ", av2mp[ii]);
		}
		printf("\n");
		}
		printf("\n");

		printf("d_PointsInThisClass.size: %d\n", d_PointsInThisClass.size());
		for (int i = 0; i < d_PointsInThisClass.size(); i++) std::cout << d_PointsInThisClass[i] << "  ";
		printf("\n\n\n");

		printf("d_CurrentUnmasked.size: %d\n", d_CurrentUnmasked.size());
		for (int i = 0; i < d_CurrentUnmasked.size(); i++) std::cout << d_CurrentUnmasked[i] << "  ";
		printf("\n\n\n");
		*/


		if (nunmasked > 0 && npoints > 0)
		{
			thrust::device_vector<float> d_X(npoints * nunmasked);
			int dimx = 32;
			int dimy = 32;
			dim3 block(dimx, dimy);
			dim3 grid(512, 512);
			matCopy << <grid, block >> > (npoints, nunmasked, nDims,
				thrust::raw_pointer_cast(&d_PointsInThisClass[0]),
				thrust::raw_pointer_cast(&d_CurrentUnmasked[0]),
				thrust::raw_pointer_cast(&d_AllVector2Mean[0]),
				thrust::raw_pointer_cast(&d_X[0]));

			//hipEvent_t start, stop;
			//float time;
			//hipEventCreate(&start);
			//hipEventCreate(&stop);

			hipblasHandle_t handle;
			hipblasCreate(&handle);
			float alpha = 1.f;
			float beta = 0.0f;

			//hipEventRecord(start, 0);

			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nunmasked, nunmasked, npoints, &alpha,
				thrust::raw_pointer_cast(d_X.data()), nunmasked, thrust::raw_pointer_cast(d_X.data()), nunmasked, &beta,
				thrust::raw_pointer_cast(d_cov.data()), nunmasked);

			//hipEventRecord(stop, 0);
			//hipEventSynchronize(stop);

			hipblasDestroy(handle);

			//hipEventElapsedTime(&time, start, stop);
			//printf("---------------------------------------------->hipblasSgemm cost times:  %f ms\n", time);
		}

		//int griddim = (nDims / BLOCKDIM) + 1;
		c_CorrectionTerm << < 1, BLOCKDIM >> > (nDims, nunmasked, nmasked, npoints,
			thrust::raw_pointer_cast(d_CurrentUnmasked.data()),
			thrust::raw_pointer_cast(d_CurrentMasked.data()),
			thrust::raw_pointer_cast(d_cov.data()),
			thrust::raw_pointer_cast(d_dig.data()),
			thrust::raw_pointer_cast(d_PointsInThisClass.data()),
			thrust::raw_pointer_cast(d_CorrectionTerm.data())
			);

		addNoiseVar << <1, BLOCKDIM >> > (nunmasked, nmasked, priorPoint,
			thrust::raw_pointer_cast(d_NoiseVariance.data()),
			thrust::raw_pointer_cast(d_CurrentUnmasked.data()),
			thrust::raw_pointer_cast(d_CurrentMasked.data()),
			thrust::raw_pointer_cast(d_cov.data()),
			thrust::raw_pointer_cast(d_dig.data())
			);

		const float factor = nClassMembers[c] + priorPoint - 1;
		thrust::transform(
			d_cov.begin(), d_cov.end(),
			thrust::make_constant_iterator((float)factor),
			d_cov.begin(),
			thrust::divides<float>());
		thrust::transform(
			d_dig.begin(), d_dig.end(),
			thrust::make_constant_iterator((float)factor),
			d_dig.begin(),
			thrust::divides<float>());
		/*
		printf("%d    %d\n", d_cov.size(), CurrentCov.Block.size());
		Output("d_cov.size: %d\n", d_cov.size());
		for (int i = 0; i < d_cov.size(); i++) std::cout << d_cov[i] << "  ";
		Output("\n\n\n");
		Output("CurrentCov.Block.size:%d\n", CurrentCov.Block.size());
		for (int i = 0; i < CurrentCov.Block.size(); i++) std::cout << CurrentCov.Block[i] << "  ";
		Output("\n\n\n");
		*/


		//thrust::copy(d_cov.begin(), d_cov.end(), CurrentCov.Block.begin());
		//thrust::copy(d_dig.begin(), d_dig.end(), CurrentCov.Diagonal.begin());

		/*
		Output("d_cov.size: %d\n", d_cov.size());
		for (int i = 0; i < d_cov.size(); i++) std::cout << d_cov[i] << "  ";
		Output("\n\n\n");
		Output("d_dig.size: %d\n", d_dig.size());
		for (int i = 0; i < d_dig.size(); i++) std::cout << d_dig[i] << "  ";
		Output("\n\n\n");

		Output("CurrentCov.Block.size:%d\n", CurrentCov.Block.size());
		for (int i = 0; i < CurrentCov.Block.size(); i++) std::cout << CurrentCov.Block[i] << "  ";
		Output("\n\n\n");
		Output("CurrentCov.Diagonal.size:%d\n", CurrentCov.Diagonal.size());
		for (int i = 0; i < CurrentCov.Diagonal.size(); i++) std::cout << CurrentCov.Diagonal[i] << "  ";
		Output("\n\n\n");

		Output("\n\n\n");
		*/



		//===============================E step=============================//

		// --- cuSOLVE input/output parameters/arrays
		int work_size = 0;
		int *devInfo;           hipMalloc(&devInfo, sizeof(int));
		// --- CUDA solver initialization
		hipsolverHandle_t solver_handle;
		hipsolverDnCreate(&solver_handle);
		hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

		// --- CUDA CHOLESKY initialization
		hipsolverDnSpotrf_bufferSize(solver_handle, uplo, nunmasked,
			thrust::raw_pointer_cast(&d_cov[0]), nunmasked, &work_size);

		// --- CUDA POTRF execution
		//float *work;   hipMalloc(&work, work_size * sizeof(float));
		thrust::device_vector<float> work(work_size);
		hipsolverDnSpotrf(solver_handle, uplo, nunmasked,
			thrust::raw_pointer_cast(&d_cov[0]), nunmasked,
			thrust::raw_pointer_cast(&work[0]), work_size, devInfo);
		int devInfo_h = 0;
		hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
		if (devInfo_h != 0) {
			Output("Unmasked E-step: Deleting class %d (%d points): covariance matrix is singular \n", (int)c, (int)NumberInClass[c]);
			ClassAlive[c] = 0;
			continue;
		}

		/*�˴�����dig���ж��Ƿ����죬������*/
		int mark = 0;
		thrust::device_vector<float> d_invdig(nmasked);
		/*
		__global__ void checkDigSingular(int nunmasked, int mark, float *d_dig, float *d_invdig) {
			int tid = blockDim.x * blockIdx.x + threadsIdx.x;
			if (tid < nmasked) {
				if (d_dig[tid] <= 0)
				{
					atomicAdd(&mark, 1);
					break;
				}
				else d_invdig[tid] = (float)sqrt(d_dig[tid]);
			}
		}*/
		checkDigSingular << <1, BLOCKDIM >> > (nunmasked, mark, thrust::raw_pointer_cast(&d_dig[0]),
			thrust::raw_pointer_cast(&d_invdig[0]));
		if (mark != 0) {
			Output("Unmasked E-step: Deleting class %d (%d points): covariance matrix is singular \n", (int)c, (int)NumberInClass[c]);
			ClassAlive[c] = 0;
			continue;
		}

		/*�˴�ͨ���Խ��߸���LogRootDet*/
		float LogRootDet = thrust::reduce(d_invdig.begin(), d_invdig.end(), (float)0, thrust::plus<float>());
		thrust::device_vector<float> d_invcov(nunmasked);
		/*
		__global__ void makeInvCov(int nunmasked, float *d_cov, float *d_invcov) {
			int tid = blockDim.x * blockIdx.x + threadsIdx.x;
			if (tid < nunmasked)
				d_invcov[tid] = d_cov[tid * nunmasked + tid];
		}*/
		makeInvCov << <1, BLOCKDIM >> > (nunmasked,
			thrust::raw_pointer_cast(&d_cov[0])
			thrust::raw_pointer_cast(&d_invcov[0]));
		LogRootDet = thrust::reduce(d_invcov.begin(), d_invcov.end(), LogRootDet, thrust::plus<float>());



		/*�˴�����cov��inverse�ĶԽ���Ԫ�أ�����invDig��*/
		thrust::device_vector<float> d_unitMat(nunmasked*nunmasked, 0.0);
		/*
		__global__ void makeUnitMat(int nunmasked, float *d_unitMat) {
			int tid = blockDim.x * blockIdx.x + threadsIdx.x;
			if (tid < nunmasked)
				d_unitMat[tid * nunmasked + tid] = 1.0;
		}
		*/
		makeUnitMat << < 1, BLOCKDIM >> > (nunmasked, thrust::raw_pointer_cast(&d_unitMat[0]));
		hipsolverDnSpotrs(solver_handle, uplo, nunmasked, nunmasked,
			thrust::raw_pointer_cast(&d_cov[0]), nunmasked,
			thrust::raw_pointer_cast(&d_unitMat[0]), nunmasked, devInfo);
		hipDeviceSynchronize();

		thrust::device_vector<float> d_invDig(nDims);
		/*
		__global__ void transInvDig(int nunmasked, int nmasked,
			float *d_unitMat, float *d_invdig,
			float *d_CurrentUnmasked, float *d_CurrenMasked,
			float *d_invDig) {
			int tid = blockDim.x * blockIdx.x + threadsIdx.x;
			if (tid < nunmasked)
				d_invDig[d_CurrentUnmasked[tid]] = d_unitMat[tid * nunmasked + tid];
			__syncthreads();
			if (tid < nmasked)
				d_invDig[d_CurrentMasked[tid]] = -1.0 / d_invdig[tid];
		}
		*/
		transInvDig << <1, BLOCKDIM >> > (nunmasked, nmasked,
			thrust::raw_pointer_cast(&d_unitMat[0]),
			thrust::raw_pointer_cast(&d_invdig[0]),
			thrust::raw_pointer_cast(&d_CurrentUnmasked[0]),
			thrust::raw_pointer_cast(&d_CurrenMasked[0]))
			//...

			/*�Ƚ�������ת������ȡ��Ҫ�����right side*/
			thrust::device_vector<float> d_points2mean(nPoints*nunmasked);
		/*
		__global__ void extractPoints2Mean(int nunmasked,int nPoints,int nDims,int clusterId,
			int *d_CurrentUnmasked,
			float *d_Data,float *d_Mean,float *d_points2mean) {
			int tidx = blockDim.x * blockIdx.x + threadIdx.x;
			int tidy = blockDim.y * blockIdx.y + threadIdx.y;
			if (tidx < nunmasked && tidy < nPoints) {
				int nd = d_CurrentUnmasked[tidx];
				d_points2mean[tidy * nunmasked + tidx] =
					d_Data[tidy * nDims + nd] - d_Mean[clusterId * nDims + nd];
			}
		}*/
		extractPoints2Mean << < (512, 512), (32, 32) >> > (nunmasked, nPoints, nDims, c,
			thrust::raw_pointer_cast(&d_CurrentUnmasked[0]),
			thrust::raw_pointer_cast(&d_Data[0]),
			thrust::raw_pointer_cast(&d_Mean[0]),
			thrust::raw_pointer_cast(&d_points2mean[0]));

		/*�˴�����������Է����飬�����ÿ��point�Ľ��ƽ����Mahal*/
		hipsolverDnSpotrs(solver_handle, uplo, nunmasked, nPoints,
			thrust::raw_pointer_cast(&d_cov[0]), nunmasked,
			thrust::raw_pointer_cast(&d_points2mean[0]), nunmasked, devInfo);
		hipDeviceSynchronize();
		hipsolverDnDestroy(solver_handle);
		//...
		float Mahal = 0;

		/*�˴�����correctionterm��invcovdig����Mahal*/
		thrust::device_vector<float> d_subMah(nDims);
		/*
		__global__ void calSubMahal(int nDims, int pId, float *d_invDig, float *d_CorrectionTerm,
			float *d_subMah) {
			int tidx = blockDim.x * blockIdx.x + threadIdx.x;
			if (tidx < nDims)
				d_subMah[tidx] = d_invDig[tidx] * d_CorrectionTerm[pId*nDims + tidx];
		}
		*/
		calSubMahal << <1, BLOCKDIM >> > (nDims, cc,
			thrust::raw_pointer_cast(&d_invDig[0]),
			thrust::raw_pointer_cast(&d_CorrectionTerm[0]),
			thrust::raw_pointer_cast(&d_subMah[0]));
		Mahal = thrust::reduce(d_subMah.begin(), d_subMah.end(), Mahal, thrust::plus<float>());

		/*�˴�����LogP*/
		d_LogP[cc*MaxPossibleClusters + c] = Mahal / 2.0
			+ LogRootDet
			- log(Weight[c])
			+ (float)(0.5*log(2 * M_PI))*nDims;
	}

	clock_t dif2 = clock() - clock2;
	clock_t dif = clock() - clock1;
	printf("---------------------------------------------->cost times:  %f ms\n", (float)dif / CLOCKS_PER_SEC);
	printf("---------------------------------------------->for loop cost times:  %f ms\n", (float)dif2 / CLOCKS_PER_SEC);
	printf("\n");
	//mtime += dif;
}