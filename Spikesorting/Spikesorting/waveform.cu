#include "hip/hip_runtime.h"


#include <stdio.h>
/*
waveform.cu:包含的函数主要是对应SpikeDetect部分的waveform的一些操作
函数作用如下：
comps_wave()：对于detect部分提取到的components，从变换后的波形data_t中提取对应的wave
normalize()：对于波形中的电位值，通过高阈值ts和低阈值tw进行归一化，方便之后计算masks和尖峰的中心时间
compute_masks():对于每一个提取到的wave，计算其masks的值
*/
/*******************************************************copy the components to the wave**************************************************************/
__global__ void comps_wave(int **wave, int s_min, int s_max, int *flit_ary, size_t num)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	if (tidx < s_max - s_min && tidy < 32)
	{
		wave[tidx][tidy] = flit_ary[(tidx + s_min) * 32 + tidy];
	}
}
/****************************************************normalize操作*************************************************************/
__global__ void normalize(float *nor_ary, float *flit_ary,float tw,float ts, size_t N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N)
	{
		if (flit_ary[tid] >= ts) nor_ary[tid] = 1;
		else if (nor_ary[tid] < tw) nor_ary[tid] = 0;
		else nor_ary[tid] = (flit_ary[tid] - tw) / (ts - tw);
	}
}
/****************************************************compute_masks操作*************************************************************/
__global__ void compute_masks(float **wave, float *mask_bin, float *masks, float tw, float ts, size_t num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < 32) masks[tid] = wave[0][tid];

	if (tid < num/32)
	for (int i = 0; i < 32; i++)
	{
		if (wave[tid][i] > masks[tid])
			masks[tid] = wave[tid][i];
	}
	__syncthreads();
	if (tid < 32)
	{
		if (mask_bin[tid] == 0)
			masks[tid] = 0;
	}
}