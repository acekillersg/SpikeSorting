#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "parameters.h"
/****************************************************abs ����*************************************************************/
__global__ void flit(float *sort_ary, size_t N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N)
	{
		if (sort_ary[tid] < 0)
			sort_ary[tid] = -sort_ary[tid];
	}
}
/*************************************************�������*********************************************************/
__device__ void swap(float &a, float &b){
	float t = a;
	a = b;
	b = t;
}
__global__ void even_sort(float *ary, int size, int *mark)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if ((tid + 1) % 2 == 1 && tid + 1 < size)
	{
		if (ary[tid] > ary[tid + 1]){
			swap(ary[tid], ary[tid + 1]);
			mark[0] = 1;
		}
	}
	__syncthreads();
}
__global__ void odd_sort(float *ary, int size, int *mark)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if ((tid + 1) % 2 == 0 && tid + 1 < size)
	{
		if (ary[tid] > ary[tid + 1]){
			swap(ary[tid], ary[tid + 1]);
			mark[1] = 1;
		}

	}
	__syncthreads();
}
/************************************************��ֵ����**************************************************************/
__global__ void Crossing(float *ary, float *sort_ary, size_t N, int *crossing)
{
	float mid = (sort_ary[N / 2 - 1] + sort_ary[N / 2]) / 2.0/0.6745;
	float high = mid * 4.5;
	float low = mid * 2.0;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid == 1) printf("%f  %f\n", high, low);
	if (tid < N)
	{
		if (ary[tid] < high && ary[tid] >= low)  crossing[tid] = 1;
		else if (ary[tid] >= high)  crossing[tid] = 2;
	}
}
/****************************************************����kernel*****************************************************************/
extern "C"
void mixGPU(int *crossing, float *ary, int *mark, size_t N)
{
	int numThreads = 1024;
	int numBlocks = (N + numThreads - 1) / numThreads;
	
	float *dev_ary = 0;
	float *dev_sort_ary = 0;
	int *dev_mark = 0;

	int *dev_crossing = 0;
	//unsigned int sharedSize = numThreads * sizeof(float);

	//hipMalloc((void**)&dev_prms, sizeof(spikedetekt_prm));

	hipMalloc((void**)&dev_ary, N*sizeof(float));
	hipMalloc((void**)&dev_sort_ary,N * sizeof(float));
	hipMalloc((void**)&dev_mark, 2 * sizeof(int));

	hipMalloc((void**)&dev_crossing, N * sizeof(int));
	hipMemset(dev_crossing, 0, N * sizeof(int));

	hipMemcpy(dev_ary, ary, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_sort_ary, ary, sizeof(float)*N, hipMemcpyHostToDevice);

	// kernel execution
	flit << <numBlocks, numThreads >> >(dev_sort_ary, N);
	while (mark[0] + mark[1] > 0)
	{
		mark[0] = 0;
		mark[1] = 0;
		hipMemcpy(dev_mark, mark, sizeof(int) * 2, hipMemcpyHostToDevice);
		even_sort << <numBlocks, numThreads >> >(dev_sort_ary, N, dev_mark);
		odd_sort << <numBlocks, numThreads >> >(dev_sort_ary, N, dev_mark);
		hipMemcpy(mark, dev_mark, 2 * sizeof(int), hipMemcpyDeviceToHost);
	}
	Crossing <<<numBlocks, numThreads >>>(dev_ary, dev_sort_ary, N, dev_crossing);


	/************************************************************CPU test******************************************************************/
	//hipMemcpy(answer, dev_answer, 2 * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(crossing, dev_crossing, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_ary);
	hipFree(dev_sort_ary);
	hipFree(dev_mark);
	hipFree(dev_crossing);
}
